#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/highgui/highgui.hpp>

/*N.B: non posso comparare un metodo di libreria con un metodo eseguito a mano quindi o comparo due metodi di libreria o scrivo sia sequenziale che kernel per
comparare.*/
//N.B al momento l'operazione da parallelizzare è l'equalizzazione e non il calcolo dell'istogramma eventualmente
//domando alla prof se devo fare pure quello. 
/* Prossima cosa da fare:
    Devo dedicarmi all'istogramma quindi l'obiettivo è [CON METODI OPENCV2]calcolare e visualizzare l'istogramma 
    dell'ultima immagine ottenuta dal preprocessing CPU, equalizzo e visualizzo l'immagine equalizzata e originale
    Dopo di questo devo scrivere il codice CPU io, faccio il passo di prima per capire qual'è l'output dell'operazione.
    Dopodiché fatto con la CPU devo scrivere codice cuda kernel per fare l'operazione di equalizzazione.
*/

cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat, float*);
cv::Mat cpu_resizeImage(cv::Mat,cv::Size, float*);
cv::Mat cpu_calcHist(cv::Mat, float*);

cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat, hipEvent_t*, float&);
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat, cv::Size size, hipEvent_t*, float&);
cv::cuda::GpuMat gpu_calcHist(cv::cuda::GpuMat ,hipEvent_t* , float& );

//cv::Mat metodoHough è l'unico che ritorna l'output finale.

int main(int argn, char *argv[]) {
    //Variables
    cv::Mat cpu_grayscaleImage, cpu_resizedImage, cpu_Hist;
    cv::cuda::GpuMat gpu_grayscaleImage, gpu_resizedImage, gpu_Hist;
    cv::Mat output; //Final output image (downloaded from GPU)
    hipEvent_t timer[2];
    cv::cuda::GpuMat gpuImage;
    float GPUelapsedTime, CPUelapsedTime;
    cv::Size size(600,600);
    //Read the input image
    cv::Mat input = cv::imread("foto.jpg");

    if (input.empty()) {
        fprintf(stderr, "Unable to load image\n");
        return -1;
    }

    //Loading of the image from the cpu to gpu
    gpuImage.upload(input);
    //Timer Evenet creation
    hipEventCreate(&timer[0]);
    hipEventCreate(&timer[1]);
    
    
    //RGB to Grayscale function (CPU)
    cpu_grayscaleImage = cpu_RGBtoGRAYSCALE(input, &CPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //RGB to Grayscale function (GPU)
    gpu_grayscaleImage = gpu_RGBtoGRAYSCALE(gpuImage, timer, GPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on GPU: %f msec\n", GPUelapsedTime);

    //Resize on CPU with GPU image as input
    cpu_resizedImage=cpu_resizeImage(cpu_grayscaleImage,size, &CPUelapsedTime);
    printf("[Resize] Execution time on CPU: %f msec\n", CPUelapsedTime);

    gpu_resizedImage= gpu_resizeImage(gpu_grayscaleImage,size, timer, GPUelapsedTime);
    printf("[Resize] Execution time on GPU: %f msec\n", GPUelapsedTime);

    cpu_Hist = cpu_calcHist(cpu_grayscaleImage, &CPUelapsedTime);
    printf("[Histogram Calculation] Execution time on CPU: %f msec\n", CPUelapsedTime);

    gpu_Hist = gpu_calcHist(gpu_resizedImage, timer, GPUelapsedTime);
    printf("[Histogram Calculation] Execution time on GPU: %f msec\n", GPUelapsedTime);
 
    //cv::imshow("Input image", input);
    //gpu_resizedImage.download(output);
    //cv::imshow("Resized and converted to grayscale image", output);
    //cv::waitKey(0);
    
    //The memory of cv::cuda::GpuMat and cv::Mat objects is automatically deallocated by the library
    hipEventDestroy(timer[0]);
    hipEventDestroy(timer[1]);
    return 0;
}


//Histogram calculation by using OpenCV routine, return the hist and not the image of the hist.
cv::Mat cpu_calcHist(cv::Mat image, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat hist, hist_image; //hist_image is the graphical rappresentation of hist.
    int histSize = 256;  // Bin's number
    float range[] = {0, 256};  // Range pixel value
    const float* histRange = {range};
    bool uniform = true;
    bool accumulate = false;
    
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::calcHist(&image, 1, 0, cv::Mat(), hist, 1, &histSize, &histRange, uniform, accumulate);
    //Histogram normalization with values between 0 and 1 with MinMax method, no masks
    cv::normalize(hist, hist, 0, 1, cv::NORM_MINMAX, -1, cv::Mat());
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return hist;
}

//Resize of the image using OpenCV (CPU)
cv::Mat cpu_resizeImage(cv::Mat in,cv::Size size, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::resize(in, out, size);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}
//Converting RGB to Grayscale using OpenCV (CPU)
cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat in, float *elapsedTime){
    struct timespec start_time, end_time;
    //Output image
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //BGR to Grayscale
    cv::cvtColor(in,out,cv::COLOR_BGR2GRAY);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//Converting RGB to Grayscale using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat gpuImage, hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    //BGR to Grayscale
    cv::cuda::cvtColor(gpuImage,out,cv::COLOR_BGR2GRAY);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);

    return out;
}

//Resize of the image using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat gpuImage, cv::Size size, hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    cv::cuda::resize(gpuImage, out, size);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);
    return out;
}

cv::cuda::GpuMat gpu_calcHist(cv::cuda::GpuMat gpuImage,hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat hist;
    //Timer's start
    hipEventRecord(timer[0], 0);
    cv::cuda::calcHist(gpuImage, hist);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);
    return hist;
}