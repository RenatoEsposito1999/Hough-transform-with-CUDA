#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <opencv2/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/highgui/highgui.hpp>
//Inserire anche l'equalizazzione usando le chiamate della libreria cuda e di opencv
//Capire se la routine openCV x CUDA usa la SM.
//Scrivere i tempi su una tabella magari grafica.

void calcCumHist(cv::Mat, int*);
void CalcCudaGrid(dim3&, dim3&, int, int);

cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat, float*);
cv::Mat cpu_resizeImage(cv::Mat,cv::Size, float*);
cv::Mat cpu_equalization(cv::Mat, int*, float*);
cv::Mat cpu_HoughTransformLine(cv::Mat, float *); 

cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat, hipEvent_t*, float&);
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat, cv::Size size, hipEvent_t*, float&);
cv::cuda::GpuMat equalizeHistOnGPU(cv::cuda::GpuMat, cv::Mat);
__global__ void equalizeHistCUDA(unsigned char*, unsigned char*,int* , int, int);
__global__ void equalizeHistCUDASM(unsigned char*, unsigned char*, int *, int , int ) ;
int main(int argn, char *argv[]){
    //Variables
    cv::Mat cpu_grayscaleImage, cpu_resizedImage, cpu_equalizedImage;
    cv::cuda::GpuMat gpuImage, gpu_grayscaleImage, gpu_resizedImage;
    int cumHist[256]={0};
    int *cumHist_device;
    dim3 nThreadPerBlocco, numBlocks;
    hipEvent_t timer[2];
    float GPUelapsedTime, CPUelapsedTime;
    cv::Size size(600,600);

    //Read the input image
    cv::Mat input = cv::imread("foto.jpg");

    if (input.empty()) {
        fprintf(stderr, "Unable to load image\n");
        return -1;
    }

    //Loading of the image from the cpu to gpu
    gpuImage.upload(input);

    //Kernel settings
    CalcCudaGrid(numBlocks,nThreadPerBlocco, size.height,size.width);
    printf("\t***Kernel settings***:\nNumber of blocks: %dx%d\tNumber of threads x bloc: %dx%d\n",numBlocks.y,numBlocks.x,nThreadPerBlocco.y, nThreadPerBlocco.x);

    //Timer Evenet creation
    hipEventCreate(&timer[0]);
    hipEventCreate(&timer[1]);
    
    //RGB to Grayscale function (CPU)
    cpu_grayscaleImage = cpu_RGBtoGRAYSCALE(input, &CPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //RGB to Grayscale function (GPU)
    gpu_grayscaleImage = gpu_RGBtoGRAYSCALE(gpuImage, timer, GPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on GPU: %f msec\n", GPUelapsedTime);

    //Resize on CPU with CPU image as input
    cpu_resizedImage=cpu_resizeImage(cpu_grayscaleImage,size, &CPUelapsedTime);
    printf("[Resize] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //Resize on GPU
    gpu_resizedImage = gpu_resizeImage(gpu_grayscaleImage,size, timer, GPUelapsedTime);
    printf("[Resize] Execution time on GPU: %f msec\n", GPUelapsedTime);
    
    //CPU Equalization by myself 
    calcCumHist(cpu_resizedImage,cumHist);
    cpu_equalizedImage = cpu_equalization(cpu_resizedImage,cumHist,&CPUelapsedTime);
    printf("[Equalization] Execution time on CPU: %f msec\n", CPUelapsedTime);

//Equalization on GPU - NO SM

    //Mem. allocation on GPU for cumHist
    hipMalloc((void**)&cumHist_device,256*sizeof(int));
    hipMemcpy(cumHist_device, cumHist, 256*sizeof(int), hipMemcpyHostToDevice);

    cv::cuda::GpuMat gpu_equalizedImage = cv::cuda::createContinuous(gpu_resizedImage.rows,gpu_resizedImage.cols,CV_8UC1);
    
    //Timer's start
    hipEventRecord(timer[0], 0);

    equalizeHistCUDA<<<numBlocks,nThreadPerBlocco>>>(gpu_resizedImage.data,gpu_equalizedImage.data,cumHist_device,gpu_resizedImage.cols,gpu_resizedImage.rows);
    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess)
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaErr));


    hipDeviceSynchronize();
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&GPUelapsedTime, timer[0], timer[1]);
    printf("[Equalization without SM] Execution time on GPU: %f msec\n", GPUelapsedTime);
    
    cv::Mat img;
    gpu_equalizedImage.download(img);
    cv::imwrite("EqualizedWithoutSM.jpg", img);
    
// END Equalization - NO SM

//Start Equalization with SM
    
    cv::cuda::GpuMat gpu_equalizedImageSM = cv::cuda::createContinuous(gpu_resizedImage.rows,gpu_resizedImage.cols,CV_8UC1);
    
    //Timer's start
    hipEventRecord(timer[0], 0);
    equalizeHistCUDASM<<<numBlocks,nThreadPerBlocco>>>(gpu_resizedImage.data,gpu_equalizedImageSM.data,cumHist_device,gpu_resizedImage.cols,gpu_resizedImage.rows);
    cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess)
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(cudaErr));
    hipDeviceSynchronize();
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&GPUelapsedTime, timer[0], timer[1]);
    printf("[Equalization with SM] Execution time on GPU: %f msec\n", GPUelapsedTime);

    cv::Mat SM;
    gpu_equalizedImageSM.download(SM);
    cv::imwrite("EqualizedWithSM.jpg", SM);
    
//END Equalization with SM


//The memory of cv::cuda::GpuMat and cv::Mat objects is automatically deallocated by the library.
//But to avoid any problem I do it manually.
    cpu_grayscaleImage.release();
    cpu_resizedImage.release();
    cpu_equalizedImage.release();
    gpuImage.release();
    gpu_grayscaleImage.release();
    gpu_resizedImage.release();
    gpu_equalizedImage.release();
    hipFree(cumHist_device);
    hipEventDestroy(timer[0]);
    hipEventDestroy(timer[1]);
    return 0;
}


//Cumulative Histogram computation
void calcCumHist(cv::Mat image, int *cumHist){
    int nBins = 256, sum=0;
    int hist[nBins];
    memset(hist,0,sizeof(hist));
    //Histogram
    for (int i = 0; i<image.rows; i++){
        for(int j = 0; j<image.cols; j++){
            unsigned char pixel_value= image.at<unsigned char>(i, j);
            hist[pixel_value]++;
        }
    }

    for (int i = 0; i<nBins; i++){
        sum+=hist[i];
        cumHist[i]=sum;
    }
}


//Histogram equalization on CPU
cv::Mat cpu_equalization(cv::Mat image,int *cumulative_hist, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat equalizedImage(cv::Size(image.rows,image.cols),CV_8UC1,cv::Scalar(255));
    int area = image.rows*image.cols, ngraylevel=256;
    uchar pixel_value;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //Equalization
    for (int i =0; i<image.rows; i++){
        for(int j = 0; j<image.cols; j++){
            pixel_value = image.at<uchar>(i,j);
            equalizedImage.at<uchar>(i,j) = ((double)ngraylevel/area)*cumulative_hist[pixel_value];
        }
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    
    return equalizedImage;
}

//Resize of the image using OpenCV (CPU)
cv::Mat cpu_resizeImage(cv::Mat in,cv::Size size, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::resize(in, out, size);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//Converting RGB to Grayscale using OpenCV (CPU)
cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat in, float *elapsedTime){
    struct timespec start_time, end_time;
    //Output image
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //BGR to Grayscale
    cv::cvtColor(in,out,cv::COLOR_BGR2GRAY);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//HoughTransform for line
cv::Mat cpu_HoughTransformLine(cv::Mat image, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat output=image.clone();

    std::vector<cv::Vec2f> lines;  // Vector for lines feature

    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::HoughLines(image, lines, 1, CV_PI / 180, 100);

    for (size_t i = 0; i < lines.size(); ++i) {
        float rho = lines[i][0];
        float theta = lines[i][1];
        cv::Point pt1, pt2;

        double a = cos(theta);
        double b = sin(theta);
        double x0 = a * rho;
        double y0 = b * rho;

        pt1.x = cvRound(x0 + 1000 * (-b));
        pt1.y = cvRound(y0 + 1000 * (a));
        pt2.x = cvRound(x0 - 1000 * (-b));
        pt2.y = cvRound(y0 - 1000 * (a));

        cv::line(output, pt1, pt2, cv::Scalar(0, 0, 255), 2, cv::LINE_AA);
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return output;
}

//Converting RGB to Grayscale using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat gpuImage, hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat out = cv::cuda::createContinuous(gpuImage.size(),gpuImage.type());
    //Timer's start
    hipEventRecord(timer[0], 0);
    //BGR to Grayscale
    cv::cuda::cvtColor(gpuImage,out,cv::COLOR_BGR2GRAY);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);

    return out;
}

//Resize of the image using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat gpuImage, cv::Size outputSize, hipEvent_t* timer, float& elapsedTime){

    cv::cuda::GpuMat out = cv::cuda::createContinuous(outputSize,gpuImage.type());
    //Timer's start
    hipEventRecord(timer[0], 0);
    cv::cuda::resize(gpuImage, out, outputSize);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);
    return out;
}

void CalcCudaGrid(dim3 &numBlocks, dim3 &nThreadPerBlocco, int rows, int cols){
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);  // 0 device's index
    //Max thread's num. x block of the gpu
    int maxThreadsPerBlock = prop.maxThreadsPerBlock;
    nThreadPerBlocco.x = min(cols, int(sqrt(maxThreadsPerBlock)));  // Max for x
    nThreadPerBlocco.y = min(rows, maxThreadsPerBlock / nThreadPerBlocco.x);  // Max for y
    numBlocks.x = (cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x;
    numBlocks.y = (rows + nThreadPerBlocco.y - 1) / nThreadPerBlocco.y;
}

__global__ void equalizeHistCUDA(unsigned char* input, unsigned char* output, int *cumulative_hist, int cols, int rows) {
    int nGrayLevels = 256, area = cols*rows;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols){
        int index = i * cols + j;
        int pixelValue = input[index];
        output[index] = static_cast<uchar>((static_cast<double>(nGrayLevels) / area) * cumulative_hist[pixelValue]);
    }
}

__global__ void equalizeHistCUDASM(unsigned char* input, unsigned char* output, int *cumulative_hist, int cols, int rows) {
    int nGrayLevels = 256, area = cols * rows;
    __shared__ int shared_cumulative_hist[256];
    int elements_per_thread = ( 256/(blockDim.x*blockDim.y) > 1 ) ? (256/blockDim.x*blockDim.y) : 1;
    int InBlockThreadID = threadIdx.x + blockDim.x * threadIdx.y; //from 0 to 1023 x block of 32x32 threads
    int start_index = InBlockThreadID * elements_per_thread;
    for (int i = 0; i < elements_per_thread; i++) {
        int index = start_index + i;
        if (index < 256)
            shared_cumulative_hist[index] = cumulative_hist[index];
    }
    __syncthreads();

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows && j < cols) {
        int index = i * cols + j;
        int pixelValue = input[index];
        output[index] = static_cast<unsigned char>((static_cast<double>(nGrayLevels) / area) * shared_cumulative_hist[pixelValue]);
    }
}

//SOLUZIONe:
// Un solo thread per blocco carica i dati nella shared memory
/*for (int i = threadIdx.x; i < 256; i += blockDim.x) {
shared_cumulative_hist[i] = cumulative_hist[i];
}
Oppure
//int elements_per_thread = 256 / (blockDim.x * blockDim.y);
    //int start_index = (blockIdx.x * blockDim.x + threadIdx.x) * elements_per_thread;
*/

