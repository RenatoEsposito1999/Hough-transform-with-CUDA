#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/highgui/highgui.hpp>
//Il problema potrebbe anche essere qualcosa che riguarda l'istogramma.




    


cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat, float*);
cv::Mat cpu_resizeImage(cv::Mat,cv::Size, float*);
cv::Mat cpu_equalization(cv::Mat, cv::Mat, float*);
cv::Mat calcHist(cv::Mat);
cv::Mat cpu_HoughTransformLine(cv::Mat, float *); //da vedere perché ritorna un output tutto nero. 

cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat, hipEvent_t*, float&);
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat, cv::Size size, hipEvent_t*, float&);
cv::cuda::GpuMat equalizeHistOnGPU(cv::cuda::GpuMat, cv::Mat);
__global__ void equalizeHistCUDA(uchar*, uchar*,float* , int, int);




int main(int argn, char *argv[]) {
    //Variables
    cv::Mat cpu_grayscaleImage, cpu_resizedImage, cpu_Hist, cpu_equalizedImage, output;
    cv::cuda::GpuMat gpu_grayscaleImage, gpu_resizedImage, gpu_Hist, gpu_equalizedImage;
    cv::Mat gpu_output; //Final output image (downloaded from GPU)
    cv::Mat cpu_output;
    cv::Mat cumHist;
    dim3 threadsBlock(16,16), numBlocks; 
    hipEvent_t timer[2];
    cv::cuda::GpuMat gpuImage;
    float GPUelapsedTime, CPUelapsedTime;
    cv::Size size(600,600);
    //Read the input image
    cv::Mat input = cv::imread("foto.jpg");

    if (input.empty()) {
        fprintf(stderr, "Unable to load image\n");
        return -1;
    }


    //Loading of the image from the cpu to gpu
    gpuImage.upload(input);
    //Timer Evenet creation
    hipEventCreate(&timer[0]);
    hipEventCreate(&timer[1]);
    
    
    //RGB to Grayscale function (CPU)
    cpu_grayscaleImage = cpu_RGBtoGRAYSCALE(input, &CPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //RGB to Grayscale function (GPU)
    gpu_grayscaleImage = gpu_RGBtoGRAYSCALE(gpuImage, timer, GPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on GPU: %f msec\n", GPUelapsedTime);

    //Resize on CPU with CPU image as input
    cpu_resizedImage=cpu_resizeImage(cpu_grayscaleImage,size, &CPUelapsedTime);
    printf("[Resize] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //Resize on GPU
    gpu_resizedImage= gpu_resizeImage(gpu_grayscaleImage,size, timer, GPUelapsedTime);
    printf("[Resize] Execution time on GPU: %f msec\n", GPUelapsedTime);

    cumHist = calcHist(cpu_resizedImage);

    //Equalization on CPU
    cpu_equalizedImage = cpu_equalization( cpu_resizedImage , cumHist, &CPUelapsedTime);
    printf("[Equalization] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //cv::imwrite("Input of Equalization.jpg", cpu_resizedImage);
    //cv::imwrite("Output_by_myself.jpg", cpu_equalizedImage);

    cv::cuda::equalizeHist(gpu_resizedImage, gpu_equalizedImage);

    gpu_equalizedImage.download(output);
    cv::imwrite("TRotuine.jpg", output);


    //EQUALIZATION ON GPU - MIA IMPLEMENTAZIONE 

    //Histogram calculation
    //Uploading of cumHist on GPU
    cv::cuda::GpuMat gpu_cumHist(cumHist);
    

    // Calcola il numero di blocchi necessari per coprire completamente l'immagine
    dim3 nThreadPerBlocco(256);
    dim3 nBlocks((gpu_resizedImage.cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x, (gpu_resizedImage.rows + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x);
    /*
        //dim3 nBlocks((gpu_resizedImage.rows * gpu_resizedImage.cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x);
        // Ricalcola il numero di blocchi necessari in base al numero effettivo di thread per blocco
        const int actualBlocksPerGridX = (imageWidth + actualThreadsPerBlock - 1) / actualThreadsPerBlock;
        const int actualBlocksPerGridY = (imageHeight + actualThreadsPerBlock - 1) / actualThreadsPerBlock;
        actualThreadsPerBlock  è nthreadperblocco;
    */
    //dim3 nBlocks((gpu_resizedImage.rows * gpu_resizedImage.cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x);
    printf("nBlocks = %d\n", nBlocks.x);
    cv::cuda::GpuMat equaliziedImgOnGPU(gpu_resizedImage.size(), gpu_resizedImage.type());
    equalizeHistCUDA<<<nBlocks, nThreadPerBlocco>>>(gpu_resizedImage.ptr<uchar>(), equaliziedImgOnGPU.ptr<uchar>(),gpu_cumHist.ptr<float>(), gpu_resizedImage.cols,  gpu_resizedImage.rows);

    cv::Mat equalized;
    equaliziedImgOnGPU.download(equalized);
    cv::imwrite("TRequalized.jpg",equalized);



    //FINE MIA IMPLEMENTAZIONE

    
    //The memory of cv::cuda::GpuMat and cv::Mat objects is automatically deallocated by the library
    hipEventDestroy(timer[0]);
    hipEventDestroy(timer[1]);
    return 0;
}


//Histogram computation - Equalized and Normalized cumulativ hist. 
cv::Mat calcHist(cv::Mat image){
    int histSize = 256;
    float sum = 0;
    //Histogram calculation
    cv::Mat hist = cv::Mat::zeros(1, histSize, CV_32F);
    for (int i = 0; i < image.rows; ++i) {
        for (int j = 0; j < image.cols; ++j) {
            int pixel_value = static_cast<int>(image.at<uchar>(i*image.cols+j));
            hist.at<float>(pixel_value)++;
        }
    }

    //Cumulative histogram
    cv::Mat cumulative_hist = cv::Mat::zeros(hist.size(), hist.type());
    for (int i = 1; i < histSize; ++i){
        sum += hist.at<float>(i);
        cumulative_hist.at<float>(i) = sum;
    }    
    //Normalization between 0-1
    //cumulative_hist /= image.total();

    return cumulative_hist;
}

//Histogram equalization on CPU
cv::Mat cpu_equalization(cv::Mat image, cv::Mat cumulative_hist, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat equalizedImage = image.clone();

    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //Equalization
    for (int i = 0; i < image.rows; ++i) {
        for (int j = 0; j < image.cols; ++j) {
            int pixel_value = static_cast<int>(image.at<uchar>(i*image.cols+j));
            equalizedImage.at<uchar>(i*image.cols+j) = cv::saturate_cast<uchar>(cumulative_hist.at<float>(pixel_value) * 255.0); //sature_cast is used to guarantee values between 0-255
        }
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    
    return equalizedImage;
}

//Resize of the image using OpenCV (CPU)
cv::Mat cpu_resizeImage(cv::Mat in,cv::Size size, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::resize(in, out, size);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//Converting RGB to Grayscale using OpenCV (CPU)
cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat in, float *elapsedTime){
    struct timespec start_time, end_time;
    //Output image
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //BGR to Grayscale
    cv::cvtColor(in,out,cv::COLOR_BGR2GRAY);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//HoughTransform for line
cv::Mat cpu_HoughTransformLine(cv::Mat image, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat output=image.clone();

    std::vector<cv::Vec2f> lines;  // Vector for lines feature

    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::HoughLines(image, lines, 1, CV_PI / 180, 100);

    for (size_t i = 0; i < lines.size(); ++i) {
        float rho = lines[i][0];
        float theta = lines[i][1];
        cv::Point pt1, pt2;

        double a = cos(theta);
        double b = sin(theta);
        double x0 = a * rho;
        double y0 = b * rho;

        pt1.x = cvRound(x0 + 1000 * (-b));
        pt1.y = cvRound(y0 + 1000 * (a));
        pt2.x = cvRound(x0 - 1000 * (-b));
        pt2.y = cvRound(y0 - 1000 * (a));

        cv::line(output, pt1, pt2, cv::Scalar(0, 0, 255), 2, cv::LINE_AA);
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return output;
}

//Converting RGB to Grayscale using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat gpuImage, hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    //BGR to Grayscale
    cv::cuda::cvtColor(gpuImage,out,cv::COLOR_BGR2GRAY);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);

    return out;
}

//Resize of the image using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat gpuImage, cv::Size size, hipEvent_t* timer, float& elapsedTime){

    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    cv::cuda::resize(gpuImage, out, size);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);
    return out;
}

__global__ void equalizeHistCUDA(uchar* data, uchar* out, float* cdf, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float scale = cdf[255];

    while (y < height) {
        while (x < width) {
            int tid = y * width + x;
            out[tid] = 1;//static_cast<uchar>(255.0 * (cdf[data[tid]] / scale));
            x += blockDim.x * gridDim.x;
        }
        x = threadIdx.x + blockIdx.x * blockDim.x;
        y += blockDim.y * gridDim.y;
    }
}

/*__global__ void equalizeHistCUDA(uchar* data, uchar* out, float* cdf, int size){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float scale = cdf[255];
    while (tid < size) {
        out[tid] = static_cast<uchar>(255.0 * (cdf[data[tid]] / scale));
        tid += blockDim.x * gridDim.x;
    }
}*/
