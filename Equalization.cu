#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <opencv2/imgproc.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include <opencv2/cudaarithm.hpp>
#include <opencv2/highgui/highgui.hpp>
//Provare anche l'approccio con la SM

/*dim3 nThreadPerBlocco(16,16);
    dim3 nBlocks((gpu_resizedImage.cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x, (gpu_resizedImage.rows + nThreadPerBlocco.y - 1) / nThreadPerBlocco.y);*/

    


cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat, float*);
cv::Mat cpu_resizeImage(cv::Mat,cv::Size, float*);
cv::Mat cpu_equalization(cv::Mat, int*, float*);
void calcCumHist(cv::Mat, int*);
cv::Mat cpu_HoughTransformLine(cv::Mat, float *); //da vedere perché ritorna un output tutto nero. 

cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat, hipEvent_t*, float&);
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat, cv::Size size, hipEvent_t*, float&);
cv::cuda::GpuMat equalizeHistOnGPU(cv::cuda::GpuMat, cv::Mat);
__global__ void equalizeHistCUDA(uchar*, uchar*,int* , int, int);




int main(int argn, char *argv[]){
    //Variables
    cv::Mat cpu_grayscaleImage, cpu_resizedImage, cpu_equalizedImage;
    cv::cuda::GpuMat gpuImage, gpu_grayscaleImage, gpu_resizedImage,gpu_equalizedImage;
    int cumHist[256]={0};
    int *cumHist_device;
    dim3 nThreadPerBlocco(32,32), numBlocks;
    hipEvent_t timer[2];
    float GPUelapsedTime, CPUelapsedTime;
    cv::Size size(600,600);



    //Read the input image
    cv::Mat input = cv::imread("foto.jpg");

    if (input.empty()) {
        fprintf(stderr, "Unable to load image\n");
        return -1;
    }


    //Loading of the image from the cpu to gpu
    gpuImage.upload(input);
    //Timer Evenet creation
    hipEventCreate(&timer[0]);
    hipEventCreate(&timer[1]);
    
    
    //RGB to Grayscale function (CPU)
    cpu_grayscaleImage = cpu_RGBtoGRAYSCALE(input, &CPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //RGB to Grayscale function (GPU)
    gpu_grayscaleImage = gpu_RGBtoGRAYSCALE(gpuImage, timer, GPUelapsedTime);
    printf("[RGB to Grayscale] Execution time on GPU: %f msec\n", GPUelapsedTime);

    //Resize on CPU with CPU image as input
    cpu_resizedImage=cpu_resizeImage(cpu_grayscaleImage,size, &CPUelapsedTime);
    printf("[Resize] Execution time on CPU: %f msec\n", CPUelapsedTime);

    //Resize on GPU
    gpu_resizedImage= gpu_resizeImage(gpu_grayscaleImage,size, timer, GPUelapsedTime);
    printf("[Resize] Execution time on GPU: %f msec\n", GPUelapsedTime);

    //CPU Equalization by myself 
    calcCumHist(cpu_resizedImage,cumHist);
    cpu_equalizedImage = cpu_equalization(cpu_resizedImage,cumHist,&CPUelapsedTime);
    printf("[Equalization] Execution time on CPU: %f msec\n", CPUelapsedTime);
    cv::imwrite("cpu_equalization.jpg",cpu_equalizedImage);


    //Equalization on GPU
    //Mem. allocation on GPU for cumHist
    hipMalloc((void**)&cumHist_device,256*sizeof(int));
    hipMemcpy(cumHist_device, cumHist, 256*sizeof(int), hipMemcpyHostToDevice);
    gpu_equalizedImage = cv::cuda::GpuMat(cv::Size(gpu_resizedImage.rows,gpu_resizedImage.cols), CV_8UC1, cv::Scalar(0));
    numBlocks.x=(gpu_resizedImage.cols + nThreadPerBlocco.x - 1) / nThreadPerBlocco.x;
    numBlocks.y=(gpu_resizedImage.rows + nThreadPerBlocco.y - 1) / nThreadPerBlocco.y;
    //equalizeHistCUDA<<<numBlocks,nThreadPerBlocco>>>(gpu_resizedImage.ptr<uchar>(),gpu_equalizedImage.ptr<uchar>(),cumHist_device,gpu_resizedImage.cols,gpu_resizedImage.rows);

    //TO DELETEEEEE
    cv::Mat test;
    gpu_equalizedImage.download(test);
    cv::imwrite("Equalization on GPU.jpg", test);
    
    
    //The memory of cv::cuda::GpuMat and cv::Mat objects is automatically deallocated by the library.
    //But to avoid any problem I do it manually.
    cpu_grayscaleImage.release();
    cpu_resizedImage.release();
    cpu_equalizedImage.release();
    gpuImage.release();
    gpu_grayscaleImage.release();
    gpu_resizedImage.release();
    gpu_equalizedImage.release();
    hipFree(cumHist_device);
    hipEventDestroy(timer[0]);
    hipEventDestroy(timer[1]);
    return 0;
}


//Cumulative Histogram computation
void calcCumHist(cv::Mat image, int *cumHist){
    int nBins = 256, sum=0;
    int hist[nBins];
    memset(hist,0,sizeof(hist));
    //Histogram
    for (int i = 0; i<image.rows; i++){
        for(int j = 0; j<image.cols; j++){
            unsigned char pixel_value= image.at<unsigned char>(i, j);
            hist[pixel_value]++;
        }
    }

    for (int i = 0; i<nBins; i++){
        sum+=hist[i];
        cumHist[i]=sum;
    }
}


//Histogram equalization on CPU
cv::Mat cpu_equalization(cv::Mat image,int *cumulative_hist, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat equalizedImage(cv::Size(image.rows,image.cols),CV_8UC1,cv::Scalar(255));
    int area = image.rows*image.cols, ngraylevel=256;
    uchar pixel_value;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //Equalization
    for (int i =0; i<image.rows; i++){
        for(int j = 0; j<image.cols; j++){
            pixel_value = image.at<uchar>(i,j);
            equalizedImage.at<uchar>(i,j) = ((double)ngraylevel/area)*cumulative_hist[pixel_value];
        }
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    
    return equalizedImage;
}

//Resize of the image using OpenCV (CPU)
cv::Mat cpu_resizeImage(cv::Mat in,cv::Size size, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::resize(in, out, size);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//Converting RGB to Grayscale using OpenCV (CPU)
cv::Mat cpu_RGBtoGRAYSCALE(cv::Mat in, float *elapsedTime){
    struct timespec start_time, end_time;
    //Output image
    cv::Mat out;
    clock_gettime(CLOCK_MONOTONIC, &start_time);
    //BGR to Grayscale
    cv::cvtColor(in,out,cv::COLOR_BGR2GRAY);
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return out;
}

//HoughTransform for line
cv::Mat cpu_HoughTransformLine(cv::Mat image, float *elapsedTime){
    struct timespec start_time, end_time;
    cv::Mat output=image.clone();

    std::vector<cv::Vec2f> lines;  // Vector for lines feature

    clock_gettime(CLOCK_MONOTONIC, &start_time);
    cv::HoughLines(image, lines, 1, CV_PI / 180, 100);

    for (size_t i = 0; i < lines.size(); ++i) {
        float rho = lines[i][0];
        float theta = lines[i][1];
        cv::Point pt1, pt2;

        double a = cos(theta);
        double b = sin(theta);
        double x0 = a * rho;
        double y0 = b * rho;

        pt1.x = cvRound(x0 + 1000 * (-b));
        pt1.y = cvRound(y0 + 1000 * (a));
        pt2.x = cvRound(x0 - 1000 * (-b));
        pt2.y = cvRound(y0 - 1000 * (a));

        cv::line(output, pt1, pt2, cv::Scalar(0, 0, 255), 2, cv::LINE_AA);
    }
    clock_gettime(CLOCK_MONOTONIC, &end_time);
    *elapsedTime = (end_time.tv_sec - start_time.tv_sec) * 1000.0 + (end_time.tv_nsec - start_time.tv_nsec) / 1000000.0;
    return output;
}

//Converting RGB to Grayscale using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_RGBtoGRAYSCALE(cv::cuda::GpuMat gpuImage, hipEvent_t* timer, float& elapsedTime){
    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    //BGR to Grayscale
    cv::cuda::cvtColor(gpuImage,out,cv::COLOR_BGR2GRAY);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);

    return out;
}

//Resize of the image using OpenCV for CUDA (GPU)
cv::cuda::GpuMat gpu_resizeImage(cv::cuda::GpuMat gpuImage, cv::Size size, hipEvent_t* timer, float& elapsedTime){

    cv::cuda::GpuMat out;
    //Timer's start
    hipEventRecord(timer[0], 0);
    cv::cuda::resize(gpuImage, out, size);
    //Timer's end
    hipEventRecord(timer[1], 0);
    hipEventSynchronize(timer[1]);
    //Elapsed time calculation
    hipEventElapsedTime(&elapsedTime, timer[0], timer[1]);
    return out;
}

__global__ void equalizeHistCUDA(uchar* input, uchar* output, int *cumulative_hist, int cols, int rows) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < rows && j < cols)
        output[i,j] = 156; 
    /*
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    float scale = cdf[255];
    while (y < rows) {
        while (x < cols) {
            int index = y * cols + x;
            out[index] = 1;//static_cast<uchar>(255.0 * (cdf[data[index]] / scale));
            x += blockDim.x * gridDim.x;
        }
        x = threadIdx.x + blockIdx.x * blockDim.x;
        y += blockDim.y * gridDim.y;
    }*/
} 


